//*************************************************************
// Assignment #1
// Name: Nodebechukwu Okoye
// GPU Date: Date of Submission (9/25/2020)
/***************************************************************
// File:     NodebechukwuOkoyeA1.cu
// Purpose:  Use CUDA to compute the product of two arrays and 
//			 store the product in another array
//           
//
// Input:    None.
// Output:   the first and last elements of the product array
//
// Compile:  sbatch NodebechukwuOkoyeA1Script
// 
//
//
// Algorithm:
//    1. Declare variable for both the host(GPU) and device(CPU)
//    2. Allocate Memory to host(CPU) and device(GPU) via malloc and cudaMalloc
//    3. Populate the arrays
//    4. Copy the given data from host to device via cudaMemCpy
//    5. Define your grid and block dimensions
//    6. Call your Kernel function
//    7. In Kernel: Generate a unique Id for array Index
//    8.	    Perform the Needed calculations
//    9. Copy the solution from device to host via cudaMemCpy
//    10. Free allocated memory via cudaFree and free
//
//*****************************************************************
//*******************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int SIZE = 4096;


__global__
void arrayMulti(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c)
{
 	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < size){
		c[i] = a[i] * b[i];
	}
	
 	
}
int main()
{
	//Variable Decleration
	 //For Host(CPU)
	unsigned long long int *A_h;
	unsigned long long int *B_h;
	unsigned long long int *C_h;
	unsigned long long int sum_h = 0;
	 //For Device(GPU)
	unsigned long long int *A_d;
	unsigned long long int *B_d;
	unsigned long long int *C_d;

	//size of memory to allocate on the device
	const int isize = SIZE*sizeof(unsigned long long int);

	//Memory Allocation
	 //for Host(CPU)
	A_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	B_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	C_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));

	 //for Device(GPU)
	hipMalloc( (void**)&A_d, isize);
 	hipMalloc( (void**)&B_d, isize);
 	hipMalloc( (void**)&C_d, isize);

	//Populate the Array
	for(int i=0; i<SIZE; i++)
	{
		A_h[i]=i;
		B_h[i]=(SIZE-1)+i;
	}

 	hipMemcpy( A_d, A_h, isize, hipMemcpyHostToDevice );
 	hipMemcpy( B_d, B_h, isize, hipMemcpyHostToDevice );


	int blockSize = 1024;
	int gridSize = (SIZE+blockSize)/blockSize;

	//Define the Block and Grid Dimension
	dim3 dimGrid(gridSize,1);
	dim3 dimBlock(blockSize,1);

	arrayMulti<<<dimGrid,dimBlock>>>(isize, A_d, B_d, C_d);

	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );
 	hipFree( A_d );
	hipFree( B_d );
	hipFree( C_d );
	

	for(int i = 0; i < SIZE; i++ ){
		sum_h += C_h[i];
	}
	
	printf("The first product is %llu\n", C_h[0]);
	printf("The last product is %llu\n", C_h[SIZE-1]);
	printf("The sum of the whole array is %llu\n", sum_h);

	free( A_h );
	free( B_h);
	free( C_h );

 	return EXIT_SUCCESS;


}