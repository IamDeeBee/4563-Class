#include "hip/hip_runtime.h"
//*************************************************************
// Assignment #1
// Name: Nodebechukwu Okoye
// GPU Date: Date of Submission (9/25/2020)
/***************************************************************
// File:     NodebechukwuOkoyeA3.cu
// Purpose:  Use CUDA to compute the product of two arrays and 
//			 store the product in another array
//           
//
// Input:    None.
// Output:   the first and last elements of the product array
//
// Compile:  sbatch NodebechukwuOkoyeA#Script
// 
//
//
// Algorithm:
//    1. Declare variable for both the host(GPU) and device(CPU)
//    2. Allocate Memory to host(CPU) and device(GPU) via malloc and hipMalloc
//    3. Populate the arrays
//    4. Copy the given data from host to device via cudaMemCpy
//    5. Define your grid and block dimensions
//    6. Call your Kernel function
//    7. In Kernel: Generate a unique Id for array Index
//    8.	    Perform the Needed calculations
//    9. Copy the solution from device to host via cudaMemCpy
//    10. Free allocated memory via hipFree and free
//
//*****************************************************************
//*******************************************************************/

#include <stdio.h>
#include <stdlib.h>

const int SIZE = 10240;




__global__
void arrayMulti2B(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c)
{
 	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
 	int jump = SIZE/5;

        for(int x = 1; x <= 5; x++){
            c[i + (jump*x)] = a[i + (jump*x)] * b[i + (jump*x)];
	}
}

__global__
void arrayMulti2BNoCyc(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c,int delta)
{
	for(int i=0; i<(delta/2);i++){
		int globIndex = (blockIdx.x*5*blockDim.x)+(i*blockDim.x+blockIdx.x*blockDim.x+threadidx.x);

		c[globIndex] = a[globIndex]*b[globIndex]
	}
}

__global__
void arrayMulti10B(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c)
{
 	int i = blockIdx.x * blockDim.x + threadIdx.x;

	c[i] = a[i] * b[i];
}

int main()
{
	//Variable Decleration
	 //For Host(CPU)
	unsigned long long int *A_h;
	unsigned long long int *B_h;
	unsigned long long int *C_h;
	 //For Device(GPU)
	unsigned long long int *A_d;
	unsigned long long int *B_d;
	unsigned long long int *C_d;

	//size of memory to allocate on the device
	const int isize = SIZE*sizeof(unsigned long long int);


	//Memory Allocation
	 //for Host(CPU)
	A_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	B_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	C_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));

	 //for Device(GPU)
	hipMalloc( (void**)&A_d, isize);
 	hipMalloc( (void**)&B_d, isize);
 	hipMalloc( (void**)&C_d, isize);

	//Populate the Array
	for(int i=0; i<SIZE; i++)
	{
		A_h[i]=2*i;
		B_h[i]=(2*i)+1;
	}

 	hipMemcpy( A_d, A_h, isize, hipMemcpyHostToDevice );
 	hipMemcpy( B_d, B_h, isize, hipMemcpyHostToDevice );
	
	
	//Block and Grid Dimension
	dim3 dimBlock1(1024,1);
	dim3 dimGrid1(2,1);
	dim3 dimGrid(10,1);
	//Define the Block and Grid Dimension for 2 blocks
	
	

	arrayMulti2B<<<dimGrid1,dimBlock1>>>(isize, A_d, B_d, C_d);
	
	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );

	printf("2 block -Cyclic(C[0] , C[1023]) = (%llu,%llu)\n", C_h[0], C_h[SIZE-1]);

	//Define the Block and Grid Dimension for 2 blocks -Not Cyclic
	int delta = SIZE/1024;
	arrayMulti2BNoCyc<<<dimGrid1,dimBlock1>>>(isize, A_d, B_d, C_d,delta);

	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );

	printf("2 block -Not Cyclic(C[0] , C[1023]) = (%llu,%llu)\n", C_h[0], C_h[SIZE-1]);
	
	//Define the Block and Grid Dimension for 10 blocks
	



	arrayMulti10B<<<dimGrid,dimBlock>>>(isize, A_d, B_d, C_d);

	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );

	printf("10 block(C[0] , C[1023]) = (%llu,%llu)\n", C_h[0], C_h[SIZE-1]);
	
 	hipFree( A_d );
	hipFree( B_d );
	hipFree( C_d );
	

	
	free( A_h );
	free( B_h);
	free( C_h );

 	return EXIT_SUCCESS;


}