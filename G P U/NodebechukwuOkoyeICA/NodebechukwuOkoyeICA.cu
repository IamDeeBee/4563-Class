//*************************************************************
// Assignment #1
// Name: Nodebechukwu Okoye
// GPU Date: Date of Submission (9/25/2020)
/***************************************************************
// File:     NodebechukwuOkoyeICA.cu
// Purpose:  Use CUDA to compute the product of two arrays and 
//			 store the product in another array
//           
//
// Input:    None.
// Output:   the first and last elements of the product array
//
// Compile:  sbatch NodebechukwuOkoyeICAScript
// 
//
//
// Algorithm:
//    1. Declare variable for both the host(GPU) and device(CPU)
//    2. Allocate Memory to host(CPU) and device(GPU) via malloc and cudaMalloc
//    3. Populate the arrays
//    4. Copy the given data from host to device via cudaMemCpy
//    5. Define your grid and block dimensions
//    6. Call your Kernel function
//    7. In Kernel: Generate a unique Id for array Index
//    8.	    Perform the Needed calculations
//    9. Copy the solution from device to host via cudaMemCpy
//    10. Free allocated memory via cudaFree and free
//
//*****************************************************************
//*******************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

const int SIZE = 10240;


__global__
void arrayMulti10B(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c)
{
 	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < size){
		c[i] = a[i] * b[i];
	}
}

__global__
void arrayMulti2B(int size ,unsigned long long int *a, unsigned long long int *b, unsigned long long int *c)
{
 	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
 	int jump = size/5;
    if(i < size){
        for(int x = 1; x <= 5; x++)
            c[i + (jump*x)] = a[i + (jump*x)] * b[i + (jump*x)];
    }
	
}

int main()
{
	//Variable Decleration
	 //For Host(CPU)
	unsigned long long int *A_h;
	unsigned long long int *B_h;
	unsigned long long int *C_h;
	 //For Device(GPU)
	unsigned long long int *A_d;
	unsigned long long int *B_d;
	unsigned long long int *C_d;

	//size of memory to allocate on the device
	const int isize = SIZE*sizeof(unsigned long long int);

	//Memory Allocation
	 //for Host(CPU)
	A_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	B_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));
	C_h = (unsigned long long int *) malloc(SIZE * sizeof(unsigned long long int ));

	 //for Device(GPU)
	hipMalloc( (void**)&A_d, isize);
 	hipMalloc( (void**)&B_d, isize);
 	hipMalloc( (void**)&C_d, isize);

	//Populate the Array
	for(int i=0; i<SIZE; i++)
	{
		A_h[i]=2*i;
		B_h[i]=(2*i)+1;
	}

 	hipMemcpy( A_d, A_h, isize, hipMemcpyHostToDevice );
 	hipMemcpy( B_d, B_h, isize, hipMemcpyHostToDevice );


	int blockSize = 1024;
	int gridSize = 10;

	//Define the Block and Grid Dimension for 2 blocks
	dim3 dimGrid1(2,1);
	dim3 dimBlock1(1024,1);

	arrayMulti2B<<<dimGrid1,dimBlock1>>>(isize, A_d, B_d, C_d);

	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );

	printf("2 block(0 , 1023) = (%llu,%llu)\n", C_h[0], C_h[SIZE-1]);

	
	//Define the Block and Grid Dimension for 10 blocks
	
	dim3 dimGrid(gridSize,1);
	dim3 dimBlock(blockSize,1);

	arrayMulti10B<<<dimGrid,dimBlock>>>(isize, A_d, B_d, C_d);

	hipMemcpy( C_h, C_d, isize, hipMemcpyDeviceToHost );

	printf("10 block(0 , 1023) = (%llu,%llu)\n", C_h[0], C_h[SIZE-1]);

 	hipFree( A_d );
	hipFree( B_d );
	hipFree( C_d );
	

	
	free( A_h );
	free( B_h);
	free( C_h );

 	return EXIT_SUCCESS;


}